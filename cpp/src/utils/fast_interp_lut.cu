#include "hip/hip_runtime.h"
#include "fast_interp_lut.hpp"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <cmath>
#include <vector>

//================================================================================
// HOST & DEVICE HELPER FUNCTIONS
//================================================================================

__host__ __device__ inline float mitchell_weight(float t) {
    const float B = 1.0f / 3.0f;
    const float C = 1.0f / 3.0f;
    float x = fabsf(t);
    float x2 = x * x;
    float x3 = x2 * x;

    if (x < 1.0f) {
        return (1.0f / 6.0f) * ((12.0f - 9.0f * B - 6.0f * C) * x3 +
                               (-18.0f + 12.0f * B + 6.0f * C) * x2 +
                               (6.0f - 2.0f * B));
    } else if (x < 2.0f) {
        return (1.0f / 6.0f) * ((-B - 6.0f * C) * x3 +
                               (6.0f * B + 30.0f * C) * x2 +
                               (-12.0f * B - 48.0f * C) * x +
                               (8.0f * B + 24.0f * C));
    }
    return 0.0f;
}

__host__ __device__ inline int safe_index(int idx, int L) {
    if (idx < 0) return -idx;
    if (idx >= L) return 2 * (L - 1) - idx;
    return idx;
}


//================================================================================
// CUDA KERNELS
//================================================================================

__global__ void apply_lut_cubic_2d_kernel(
    float* output,
    const float* image,
    const float* lut,
    int height, int width,
    int lut_size, int lut_channels)
{
    const int j = blockIdx.x * blockDim.x + threadIdx.x; // width index
    const int i = blockIdx.y * blockDim.y + threadIdx.y; // height index

    if (i >= height || j >= width) return;

    const int image_pixel_idx = (i * width + j);
    const float x_in = image[image_pixel_idx * 2 + 0] * (lut_size - 1);
    const float y_in = image[image_pixel_idx * 2 + 1] * (lut_size - 1);

    const int x_base = floorf(x_in), y_base = floorf(y_in);
    const float x_frac = x_in - x_base, y_frac = y_in - y_base;

    float wx[4], wy[4];
    wx[0] = mitchell_weight(x_frac + 1.0f); wx[1] = mitchell_weight(x_frac);
    wx[2] = mitchell_weight(x_frac - 1.0f); wx[3] = mitchell_weight(x_frac - 2.0f);
    wy[0] = mitchell_weight(y_frac + 1.0f); wy[1] = mitchell_weight(y_frac);
    wy[2] = mitchell_weight(y_frac - 1.0f); wy[3] = mitchell_weight(y_frac - 2.0f);

    float weight_sum = 0.0f;
    float out_val[16] = {0.0f}; // Max channels supported

    for (int m = 0; m < 4; ++m) {
        int y_idx = safe_index(y_base - 1 + m, lut_size);
        for (int n = 0; n < 4; ++n) {
            int x_idx = safe_index(x_base - 1 + n, lut_size);
            float weight = wx[n] * wy[m];
            weight_sum += weight;
            // Index into flattened LUT: [x*L + y, channel] for row-major order
            int lut_pixel_idx = x_idx * lut_size + y_idx;
            for (int c = 0; c < lut_channels; ++c) {
                out_val[c] += weight * lut[lut_pixel_idx * lut_channels + c];
            }
        }
    }

    const int output_pixel_idx = (i * width + j);
    if (weight_sum != 0.0f) {
        for (int c = 0; c < lut_channels; ++c) {
            output[output_pixel_idx * lut_channels + c] = out_val[c] / weight_sum;
        }
    }
}

__global__ void apply_lut_cubic_3d_kernel(
    float* output,
    const float* image,
    const float* lut,
    int height, int width,
    int lut_size, int lut_channels)
{
    const int j = blockIdx.x * blockDim.x + threadIdx.x;
    const int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= height || j >= width) return;

    const int image_pixel_idx = (i * width + j);
    const float r_in = image[image_pixel_idx * 3 + 0] * (lut_size - 1);
    const float g_in = image[image_pixel_idx * 3 + 1] * (lut_size - 1);
    const float b_in = image[image_pixel_idx * 3 + 2] * (lut_size - 1);

    const int r_base = floorf(r_in), g_base = floorf(g_in), b_base = floorf(b_in);
    const float r_frac = r_in - r_base, g_frac = g_in - g_base, b_frac = b_in - b_base;

    float wr[4], wg[4], wb[4];
    wr[0] = mitchell_weight(r_frac + 1); wr[1] = mitchell_weight(r_frac);
    wr[2] = mitchell_weight(r_frac - 1); wr[3] = mitchell_weight(r_frac - 2);
    wg[0] = mitchell_weight(g_frac + 1); wg[1] = mitchell_weight(g_frac);
    wg[2] = mitchell_weight(g_frac - 1); wg[3] = mitchell_weight(g_frac - 2);
    wb[0] = mitchell_weight(b_frac + 1); wb[1] = mitchell_weight(b_frac);
    wb[2] = mitchell_weight(b_frac - 1); wb[3] = mitchell_weight(b_frac - 2);

    float weight_sum = 0.0f;
    float out_val[16] = {0.0f};

    for (int m = 0; m < 4; ++m) {
        int r_idx = safe_index(r_base - 1 + m, lut_size);
        for (int n = 0; n < 4; ++n) {
            int g_idx = safe_index(g_base - 1 + n, lut_size);
            for (int p = 0; p < 4; ++p) {
                int b_idx = safe_index(b_base - 1 + p, lut_size);
                float weight = wr[m] * wg[n] * wb[p];
                weight_sum += weight;
                int lut_pixel_idx = (r_idx * lut_size + g_idx) * lut_size + b_idx;
                for (int c = 0; c < lut_channels; ++c) {
                    out_val[c] += weight * lut[lut_pixel_idx * lut_channels + c];
                }
            }
        }
    }

    const int output_pixel_idx = (i * width + j);
    if (weight_sum != 0.0f) {
        for (int c = 0; c < lut_channels; ++c) {
            output[output_pixel_idx * lut_channels + c] = out_val[c] / weight_sum;
        }
    }
}


//================================================================================
// HOST-FACING FUNCTIONS
//================================================================================

namespace agx {

std::vector<float> cubic_interp_lut_at_2d(const nc::NdArray<float>& lut, float x, float y) {
    // The LUT is flattened as [L*L, C] where L is the size of each dimension
    const int L = static_cast<int>(round(sqrt(lut.shape().rows)));
    const int channels = lut.shape().cols;
    const int x_base = floorf(x), y_base = floorf(y);
    const float x_frac = x - x_base, y_frac = y - y_base;

    float wx[4], wy[4];
    wx[0] = mitchell_weight(x_frac + 1); wx[1] = mitchell_weight(x_frac);
    wx[2] = mitchell_weight(x_frac - 1); wx[3] = mitchell_weight(x_frac - 2);
    wy[0] = mitchell_weight(y_frac + 1); wy[1] = mitchell_weight(y_frac);
    wy[2] = mitchell_weight(y_frac - 1); wy[3] = mitchell_weight(y_frac - 2);

    float weight_sum = 0.0f;
    std::vector<float> out(channels, 0.0f);

    for (int i = 0; i < 4; ++i) {
        int xi = safe_index(x_base - 1 + i, L);
        for (int j = 0; j < 4; ++j) {
            int yj = safe_index(y_base - 1 + j, L);
            float weight = wx[i] * wy[j];
            weight_sum += weight;
            // Index into flattened LUT: [x*L + y, channel] for row-major order
            int lut_idx = xi * L + yj;
            for (int c = 0; c < channels; ++c) {
                out[c] += weight * lut(lut_idx, c);
            }
        }
    }

    if (weight_sum != 0.0f) {
        for (int c = 0; c < channels; ++c) out[c] /= weight_sum;
    }
    return out;
}

std::vector<float> cubic_interp_lut_at_3d(const nc::NdArray<float>& lut, float r, float g, float b) {
    // The LUT is flattened as [L*L*L, C] where L is the size of each dimension
    const int L = static_cast<int>(round(cbrt(lut.shape().rows)));
    const int channels = lut.shape().cols;
    const int r_base = floorf(r), g_base = floorf(g), b_base = floorf(b);
    const float r_frac = r - r_base, g_frac = g - g_base, b_frac = b - b_base;

    float wr[4], wg[4], wb[4];
    wr[0] = mitchell_weight(r_frac + 1); wr[1] = mitchell_weight(r_frac);
    wr[2] = mitchell_weight(r_frac - 1); wr[3] = mitchell_weight(r_frac - 2);
    wg[0] = mitchell_weight(g_frac + 1); wg[1] = mitchell_weight(g_frac);
    wg[2] = mitchell_weight(g_frac - 1); wg[3] = mitchell_weight(g_frac - 2);
    wb[0] = mitchell_weight(b_frac + 1); wb[1] = mitchell_weight(b_frac);
    wb[2] = mitchell_weight(b_frac - 1); wb[3] = mitchell_weight(b_frac - 2);

    float weight_sum = 0.0f;
    std::vector<float> out(channels, 0.0f);

    for (int i = 0; i < 4; ++i) {
        int ri = safe_index(r_base - 1 + i, L);
        for (int j = 0; j < 4; ++j) {
            int gj = safe_index(g_base - 1 + j, L);
            for (int k = 0; k < 4; ++k) {
                int bk = safe_index(b_base - 1 + k, L);
                float weight = wr[i] * wg[j] * wb[k];
                weight_sum += weight;
                // Index into flattened LUT: [(r*L + g)*L + b, channel]
                int lut_idx = (ri * L + gj) * L + bk;
                for (int c = 0; c < channels; ++c) {
                    out[c] += weight * lut(lut_idx, c);
                }
            }
        }
    }

    if (weight_sum != 0.0f) {
        for (int c = 0; c < channels; ++c) out[c] /= weight_sum;
    }
    return out;
}

nc::NdArray<float> apply_lut_cubic_2d(const nc::NdArray<float>& lut, const nc::NdArray<float>& image, int height, int width) {
    const int lut_size = static_cast<int>(round(sqrt(lut.shape().rows)));
    const int lut_channels = lut.shape().cols;
    auto output = nc::NdArray<float>(height * width, lut_channels);

    float *dev_lut, *dev_image, *dev_output;
    hipMalloc(&dev_lut, lut.nbytes());
    hipMalloc(&dev_image, image.nbytes());
    hipMalloc(&dev_output, output.nbytes());

    hipMemcpy(dev_lut, lut.data(), lut.nbytes(), hipMemcpyHostToDevice);
    hipMemcpy(dev_image, image.data(), image.nbytes(), hipMemcpyHostToDevice);

    const dim3 threadsPerBlock(16, 16);
    const dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    apply_lut_cubic_2d_kernel<<<numBlocks, threadsPerBlock>>>(dev_output, dev_image, dev_lut, height, width, lut_size, lut_channels);
    
    hipDeviceSynchronize();
    if (hipGetLastError() != hipSuccess) throw std::runtime_error("CUDA Kernel Launch Error");

    hipMemcpy(output.data(), dev_output, output.nbytes(), hipMemcpyDeviceToHost);

    hipFree(dev_lut); hipFree(dev_image); hipFree(dev_output);
    return output;
}

nc::NdArray<float> apply_lut_cubic_3d(const nc::NdArray<float>& lut, const nc::NdArray<float>& image, int height, int width) {
    const int lut_size = static_cast<int>(round(cbrt(lut.shape().rows)));
    const int lut_channels = lut.shape().cols;
    auto output = nc::NdArray<float>(height * width, lut_channels);

    float *dev_lut, *dev_image, *dev_output;
    hipMalloc(&dev_lut, lut.nbytes());
    hipMalloc(&dev_image, image.nbytes());
    hipMalloc(&dev_output, output.nbytes());

    hipMemcpy(dev_lut, lut.data(), lut.nbytes(), hipMemcpyHostToDevice);
    hipMemcpy(dev_image, image.data(), image.nbytes(), hipMemcpyHostToDevice);

    const dim3 threadsPerBlock(16, 16);
    const dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    apply_lut_cubic_3d_kernel<<<numBlocks, threadsPerBlock>>>(dev_output, dev_image, dev_lut, height, width, lut_size, lut_channels);

    hipDeviceSynchronize();
    if (hipGetLastError() != hipSuccess) throw std::runtime_error("CUDA Kernel Launch Error");

    hipMemcpy(output.data(), dev_output, output.nbytes(), hipMemcpyDeviceToHost);

    hipFree(dev_lut); hipFree(dev_image); hipFree(dev_output);
    return output;
}

} // namespace agx
